#include "hip/hip_runtime.h"
 
#include <stdio.h>
#include <fcntl.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <stdint.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include "common.h"
#include "VisualCryptographyGPU.h"
 
__constant__ level WhiteShare1_d[2][2] = {0,0,1,1};
__constant__ level WhiteShare2_d[2][2] = {0,0,1,1};
__constant__ level BlackShare1_d[2][2] = {0,0,1,1};
__constant__ level BlackShare2_d[2][2] = {1,1,0,0};

void CheckCUDAError(const char *msg)
{
     hipError_t code =hipGetLastError();
      if(code!=hipSuccess)
      {
          fprintf(stderr,"Cuda Error: %s: %s.\n",msg,hipGetErrorString(code));
          exit(EXIT_FAILURE);
      }
}

__global__ void CodecKernel(level *pImage_d,level *pShare1_d, level *pShare2_d, int iWidth, int iHeight, int iCodecPath)
{
	
// ********************************************************************************
// DO NOT CHANGE ANYTHING BEFORE THIS POINT in YOUR CODE                          *
// Your CUDA Kernel should go here.                                               *
//   										  									  *
// 1) If iCodecPath=ENCODE, the code performs encoding. In that case              *
//    input image of size (iHeight x iWidth) pixels of (type level) stored at GPU *
//    address pImage_d. The share images of size (2*iHeight x 2*iWidth) are stored*
//    at GPU addresses pShare1_d and pShare2_d.                                   *
//            																	  *
// 2) If iCodecPath=DECODE, the code performs decoding. In that case              *
//    output image of size (iHeight x iWidth) pixels of (type level) is stored    *
//    at GPU address pImage_d. The share images of size (iHeight x iWidth) are    *
//    stored at GPU addresses pShare1_d and pShare2_d.                            *
// ********************************************************************************
//                                        ^
//                                        |
//                                        |
	int pos_x = (blockIdx.x * blockDim.x) + threadIdx.x;
        int pos_y = (blockIdx.y * blockDim.y) + threadIdx.y;
	//int posx = pos_x;
	int pos = pos_y ;
	int j,r;
	//int i,q;
	int Width = 2*iWidth;
	//int Height = 2*iHeight;
	int x;
	//int WhiteShare1[4];
	//int WhiteShare2[4];
	//int BlackShare1[4];
	//int BlackShare2[4];

///////////////Random Number Generation%%%%%%%%%%%%%%%%%
	int seed = pos;
///////////////////////////////////////////////////////

//////////////Create Shares///////////////////////////
//////////////////////////////////////////////////////	
  	if(iCodecPath == ENCODE)
        {
//	j = igned int seed = thread_id;
	
	if(pos<iHeight)
	{
	//if(i<=iWidth)
	
	for(j=0;j<iWidth;j++)
	{
	 hiprandState s;
        hiprand_init(seed, 0, 0, &s);
        x = (hiprand(&s)%2);

        if(pImage_d[pos*iWidth+j] ==WHITE)
	{
	if(x==0)
	{
	pShare1_d[2*pos*Width+(2*j)] = WhiteShare1_d[0][0];
	pShare1_d[2*pos*Width+(2*j)+1] = WhiteShare1_d[1][0];  
	pShare1_d[2*pos*Width+(2*j)+(Width)] = WhiteShare1_d[1][1];
	pShare1_d[2*pos*Width+(2*j)+(Width)+1] = WhiteShare1_d[0][1];

	pShare2_d[2*pos*Width+(2*j)] = WhiteShare2_d[0][0];
        pShare2_d[2*pos*Width+(2*j)+1] = WhiteShare2_d[1][0];
        pShare2_d[2*pos*Width+(2*j)+(Width)] = WhiteShare2_d[1][1];
        pShare2_d[2*pos*Width+(2*j)+(Width)+1] = WhiteShare2_d[0][1];
	}
	else
	{
	pShare1_d[2*pos*Width+(2*j)] = WhiteShare2_d[0][0];
        pShare1_d[2*pos*Width+(2*j)+1] = WhiteShare2_d[1][0];
        pShare1_d[2*pos*Width+(2*j)+(Width)] = WhiteShare2_d[1][1];
        pShare1_d[2*pos*Width+(2*j)+(Width)+1] = WhiteShare2_d[0][1];

        pShare2_d[2*pos*Width+(2*j)] = WhiteShare1_d[0][0];
        pShare2_d[2*pos*Width+(2*j)+1] = WhiteShare1_d[1][0];
        pShare2_d[2*pos*Width+(2*j)+(Width)] = WhiteShare1_d[1][1];
        pShare2_d[2*pos*Width+(2*j)+(Width)+1] = WhiteShare1_d[0][1];
	}
	}
	else 
	{
	if(x==0)
	{
        pShare1_d[2*pos*Width+(2*j)] = BlackShare1_d[0][0];
        pShare1_d[2*pos*Width+(2*j)+1] =BlackShare1_d[1][0];
        pShare1_d[2*pos*Width+(2*j)+(Width)] = BlackShare1_d[1][1];
        pShare1_d[2*pos*Width+(2*j)+(Width)+1] = BlackShare1_d[0][1];
        
        pShare2_d[2*pos*Width+(2*j)] = BlackShare2_d[0][0];
        pShare2_d[2*pos*Width+(2*j)+1] =BlackShare2_d[1][0];
        pShare2_d[2*pos*Width+(2*j)+(Width)] = BlackShare2_d[1][1];
        pShare2_d[2*pos*Width+(2*j)+(Width)+1] = BlackShare2_d[0][1];
	}
	else
	{
	pShare1_d[2*pos*Width+(2*j)] = BlackShare2_d[0][0];
        pShare1_d[2*pos*Width+(2*j)+1] =BlackShare2_d[1][0];
        pShare1_d[2*pos*Width+(2*j)+(Width)] = BlackShare2_d[1][1];
        pShare1_d[2*pos*Width+(2*j)+(Width)+1] = BlackShare2_d[0][1];

        pShare2_d[2*pos*Width+(2*j)] = BlackShare1_d[0][0];
        pShare2_d[2*pos*Width+(2*j)+1] =BlackShare1_d[1][0];
        pShare2_d[2*pos*Width+(2*j)+(Width)] = BlackShare1_d[1][1];
        pShare2_d[2*pos*Width+(2*j)+(Width)+1] = BlackShare1_d[0][1];

	}
	
	}
	//j=j+2;
	}	
	}
 	}
	
//*************Decode***************//
	if(iCodecPath == DECODE)
        {
//      j = 0;
        if(pos<iHeight)
        {
        for(r=0;r<iWidth;r++)
        {
	//pImage_d[i*Width+j] = pShare1_d[i*Width+j] & pShare2_d[i*Width+j];
	pImage_d[pos*iWidth+r]= pShare1_d[pos*iWidth+r] & pShare2_d[pos*iWidth+r] ;
	}
	}
	}

// ********************************************************************************
	

// ********************************************************************************
// Your CUDA code ends here.                                                      *                                                      *
// DO NOT CHANGE ANYTHING AFTER THIS POINT in YOUR CODE                           *
// ******************************************************************************

}

void VCEncoderGPU(ImageData *pcShare1, ImageData *pcShare2, ImageData *pcImageData, TimeRecord *pTR)
{
 	level *pShare1, *pShare2;
	level *pShare1_d, *pShare2_d;
	level *pImage_d;
	struct timeval start,stop;
	int blockSizeX, blockSizeY, gridSizeX, gridSizeY;
	
	printf("GPU Encoding... \n");
	
	//----------Fill in shares----------//	
	pcShare1->iWidth  = 2 * pcImageData->iWidth;
	pcShare1->iHeight = 2 * pcImageData->iHeight;
	pcShare2->iWidth  = 2 * pcImageData->iWidth;
	pcShare2->iHeight = 2 * pcImageData->iHeight;
    hipDeviceReset();
	//Fill in file header
	memcpy(&(pcShare1->cBmpFH), &(pcImageData->cBmpFH), sizeof(BitMapFileHeader));
	memcpy(&(pcShare2->cBmpFH), &(pcImageData->cBmpFH), sizeof(BitMapFileHeader));
	//Fill in info header
	memcpy(&(pcShare1->cBmpIH), &(pcImageData->cBmpIH), sizeof(BitMapInfoHeader));
	memcpy(&(pcShare2->cBmpIH), &(pcImageData->cBmpIH), sizeof(BitMapInfoHeader));
	//Fill in color table
	memcpy(pcShare1->cBmpImage, pcImageData->cBmpImage, 2*sizeof(BitMapImage));
	memcpy(pcShare2->cBmpImage, pcImageData->cBmpImage, 2*sizeof(BitMapImage));

	//----------Adjust shares----------//
	sprintf(pcShare1->imageName, "Share1G_%s",pcImageData->imageName); //adjust name
	sprintf(pcShare2->imageName, "Share2G_%s",pcImageData->imageName);
	pcShare1->cBmpIH.width  = pcShare1->iWidth; //adjust width
	pcShare2->cBmpIH.width  = pcShare2->iWidth;
	pcShare1->cBmpIH.height = pcShare1->iHeight;  //adjust height
	pcShare2->cBmpIH.height = pcShare2->iHeight;  
	pcShare1->cBmpIH.biSizeImage = pcShare1->cBmpIH.height * (((pcShare1->cBmpIH.bitPix * pcShare1->cBmpIH.width + 31) / 32) * 4); //adjust image size
	pcShare2->cBmpIH.biSizeImage = pcShare2->cBmpIH.height * (((pcShare2->cBmpIH.bitPix * pcShare2->cBmpIH.width + 31) / 32) * 4); //adjust image size	
	pcShare1->cBmpFH.bfSize = pcShare1->cBmpIH.biSizeImage + pcShare1->cBmpFH.bfOffBits;
	pcShare2->cBmpFH.bfSize = pcShare2->cBmpIH.biSizeImage + pcShare2->cBmpFH.bfOffBits;

	//------Generate shares pixels---------//
	pcShare1->imgData = (level *)malloc(4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	pcShare2->imgData = (level *)malloc(4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	pShare1 = pcShare1->imgData;
	pShare2 = pcShare2->imgData;
	 	
	gettimeofday(&start,0);

	//------GPU Memory Preparation-------//
	hipMalloc( (void**)&pImage_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Original Image GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare1_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share1 GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare2_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share2 GPU Memory Allocation Failed");
	
	//-------Transfer orignal image-------//
	hipMemcpy(pImage_d, pcImageData->imgData, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyHostToDevice);
	CheckCUDAError("Copy Original Image to GPU Failed");
 
	gettimeofday(&stop,0);
	pTR->MemTransferTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
	
	gettimeofday(&start,0);
	//-----GPU Kernel Launch-----//
// ********************************************************************************
// DO NOT CHANGE ANYTHING BEFORE THIS POINT in YOUR CODE                          *
// Your CUDA block size and grid size parameters go in here.                      *
// ********************************************************************************
	//Fill in here 
	blockSizeX = 1;
	blockSizeY = 1024;
	gridSizeX  = 1;
	gridSizeY  = 8;
// ********************************************************************************
// End of CUDA block size and grid size parameters                                *
// DO NOT CHANGE ANYTHING AFTER THIS POINT in YOUR CODE                           *
// ********************************************************************************	

	printf("|--Block Config: %d x %d\n",blockSizeX,blockSizeY);
	printf("|--Grid  Config: %d x %d\n",gridSizeX,gridSizeY);	
	dim3 blocksInGrid(gridSizeX,gridSizeY);
	dim3 threadsInBlock(blockSizeX,blockSizeY);
	CodecKernel<<<blocksInGrid, threadsInBlock>>>(pImage_d,pShare1_d,pShare2_d,pcImageData->iWidth, pcImageData->iHeight, ENCODE);
	hipDeviceSynchronize();
	CheckCUDAError("Encryption Kernel Failed");
	gettimeofday(&stop,0);
	pTR->EncryptionTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	gettimeofday(&start,0);
	
	//------Transfer back shares------//
	hipMemcpy(pShare1,pShare1_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy Share1 to CPU Failed");
	hipMemcpy(pShare2,pShare2_d, 4 * pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy Share2 to CPU Failed");

	gettimeofday(&stop,0);
	pTR->MemTransferTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;

	hipFree(pImage_d);
	hipFree(pShare1_d);
	hipFree(pShare2_d);
}

void VCDecoderGPU(ImageData *pcShare1, ImageData *pcShare2, char *pInputImageName, ImageData *pcImageData, TimeRecord *pTR)
{
 	level *pShare1, *pShare2;
	level *pShare1_d, *pShare2_d;
	level *pImage_d;
	struct timeval start,stop;
	int blockSizeX, blockSizeY, gridSizeX, gridSizeY;
	
	printf("GPU Decoding ...\n");
	pShare1 = pcShare1->imgData;
	pShare2 = pcShare2->imgData;
    hipDeviceReset();
     
	//------GPU Memory Preparation-------//	
	memcpy(pcImageData, pcShare1, sizeof(ImageData));
	pcImageData->imgData = (level *)malloc(pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	sprintf(pcImageData->imageName, "ReconG_%s",pInputImageName);
	//Memory Allocation 
	hipMalloc( (void**)&pImage_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Reconstructed Image GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare1_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share1 GPU Memory Allocation Failed");
	hipMalloc( (void**)&pShare2_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level));
	CheckCUDAError("Share2 GPU Memory Allocation Failed");
	
	//Transfer shares
	gettimeofday(&start,0);
	hipMemcpy(pShare1_d, pShare1, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyHostToDevice);
	CheckCUDAError("Copy Share1 to GPU Failed");
	hipMemcpy(pShare2_d, pShare2, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyHostToDevice);
	CheckCUDAError("Copy Share2 to GPU Failed");
	gettimeofday(&stop,0);
	pTR->MemTransferTimeDecode += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
	
// ********************************************************************************
	//Fill in here 
	blockSizeX = 1;
	blockSizeY = 1024;
	gridSizeX  = 1;
	gridSizeY  = 8;
// ********************************************************************************
// End of CUDA block size and grid size parameters                                *
// DO NOT CHANGE ANYTHING AFTER THIS POINT in YOUR CODE                           *
// ********************************************************************************	
	printf("|--Block Config: %d x %d\n",blockSizeX,blockSizeY);
	printf("|--Grid  Config: %d x %d\n",gridSizeX,gridSizeY);	
	dim3 blocksInGrid(gridSizeX,gridSizeY);
	dim3 threadsInBlock(blockSizeX,blockSizeY);
	CodecKernel<<<blocksInGrid, threadsInBlock>>>(pImage_d,pShare1_d,pShare2_d,pcImageData->iWidth, pcImageData->iHeight, DECODE);
	hipDeviceSynchronize();
	CheckCUDAError("Decryption Kernel Failed");
	gettimeofday(&stop,0);
	pTR->DecodeTime += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
		
	//------Transfer back reconstructed image------//
	gettimeofday(&start,0);
	hipMemcpy(pcImageData->imgData,pImage_d, pcImageData->iHeight * pcImageData->iWidth * sizeof(level), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy Reconstructed image to CPU Failed");
	gettimeofday(&stop,0);
	pTR->MemTransferTimeDecode += ((stop.tv_sec - start.tv_sec) * 1000000 + (stop.tv_usec - start.tv_usec)) / 1000;
	
	//----------Free memory----------//
 	hipFree(pImage_d);
	hipFree(pShare1_d);
	hipFree(pShare2_d);
	
}


 



